#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/cluster_launch.hpp"
#include "cutlass/arch/barrier.h"
#include "cutlass/pipeline/sm90_pipeline.hpp"

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"
#include "cutlass/arch/mma_sm90.h"
#include "cutlass/device_kernel.h"

#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_conversion.h>
#include <cutlass/numeric_types.h>

using namespace cute;

template <typename To_type, typename Engine, typename Layout>
__forceinline__ __device__ auto convert_type(Tensor<Engine, Layout> const &tensor) {
    using From_type = typename Engine::value_type;
    constexpr int numel = decltype(size(tensor))::value;
    cutlass::NumericArrayConverter<To_type, From_type, numel> convert_op;
    // HACK: this requires tensor to be "contiguous"
    auto frag = convert_op(*reinterpret_cast<const cutlass::Array<From_type, numel> *>(tensor.data()));
    return make_tensor(make_rmem_ptr<To_type>(&frag), tensor.layout());
    // Tensor out = make_tensor_like<To_type>(tensor);
    // cute::copy(make_tensor(make_rmem_ptr<To_type>(&frag), tensor.layout()), out);
    // return out;
}

template <class ElementA,
          class ElementB,
          class ElementC,
          class SmemLayoutA,  // (M,K,P)
          class SmemLayoutB,  // (N,K,P)
          class SmemLayoutC>  
struct SharedStorage
{
  array_aligned<ElementA, cosize_v<SmemLayoutA>> smem_A;
  // put in smem_C
  union {
    array_aligned<ElementB, cosize_v<SmemLayoutB>> smem_B;
    array_aligned<ElementC, cosize_v<SmemLayoutC>> smem_C;
  };

  uint64_t tma_barrier[size<2>(SmemLayoutA{})]; // size = P
  uint64_t mma_barrier[size<2>(SmemLayoutA{})];

  
};

template <class ProblemShape, class CtaTiler,
          class TA, class SmemLayoutA, class TmaA,
          class TB, class SmemLayoutB, class TmaB,
          class TC, class SmemLayoutC, class TmaC,
          class TiledMma>
__global__ static
__launch_bounds__(decltype(size(TiledMma{}))::value)
void
gemm_device(ProblemShape shape_MNK, CtaTiler cta_tiler,
            TA const* A, CUTLASS_GRID_CONSTANT TmaA const tma_a,
            TB const* B, CUTLASS_GRID_CONSTANT TmaB const tma_b,
            TC      * C, CUTLASS_GRID_CONSTANT TmaC const tma_store,
            TiledMma mma)
{   
  
  // Preconditions
  CUTE_STATIC_ASSERT_V(rank(shape_MNK) == Int<3>{});                   // (M, N, K)
  CUTE_STATIC_ASSERT_V(rank(cta_tiler) == Int<3>{});                   // (BLK_M, BLK_N, BLK_K)

  static_assert(is_static<SmemLayoutA>::value);
  static_assert(is_static<SmemLayoutB>::value);

  CUTE_STATIC_ASSERT_V(size<0>(SmemLayoutA{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<0>(SmemLayoutB{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(SmemLayoutA{}) == size<2>(cta_tiler));  // BLK_K
  CUTE_STATIC_ASSERT_V(size<1>(SmemLayoutB{}) == size<2>(cta_tiler));  // BLK_K

  //
  // Full and Tiled Tensors
  //

  // Represent the full tensors
  auto [M, N, K] = shape_MNK;
  Tensor mA = tma_a.get_tma_tensor(make_shape(M,K));                   // (M,K) TMA Tensor
  Tensor mB = tma_b.get_tma_tensor(make_shape(N,K));                   // (N,K) TMA Tensor
//   Tensor mC = make_tensor(make_gmem_ptr(C), make_shape(M,N), dC);      // (M,N)

  // Get the appropriate blocks for this thread block
  auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _);              // (m,n,k)
  Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X,_1>{});  // (BLK_M,BLK_K,k)
  Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step< X,_1,_1>{});  // (BLK_N,BLK_K,k)
//   Tensor gC = local_tile(mC, cta_tiler, cta_coord, Step<_1,_1, X>{});  // (BLK_M,BLK_N)

  // Shared memory tensors
  extern __shared__ char shared_memory[];
  using SharedStorage = SharedStorage<TA, TB, TC, SmemLayoutA, SmemLayoutB, SmemLayoutC>;
  SharedStorage& smem = *reinterpret_cast<SharedStorage*>(shared_memory);
  Tensor sA = make_tensor(make_smem_ptr(smem.smem_A.data()), SmemLayoutA{}); // (BLK_M,BLK_K,PIPE)
  Tensor sB = make_tensor(make_smem_ptr(smem.smem_B.data()), SmemLayoutB{}); // (BLK_N,BLK_K,PIPE)

  //
  // Partition the copying of A and B tiles
  //
  // TUTORIAL:
  //   These are TMA partitionings, which have a dedicated custom partitioner.
  //   The Int<0>, Layout<_1> indicates that the TMAs are not multicasted.
  //     Any multicasting must be in conformance with tma_x constructed with make_tma_atom on host.
  //   The group_modes<0,2> transforms the (X,Y,Z)-shaped tensors into ((X,Y),Z)-shaped tensors
  //     with the understanding that the TMA is responsible for everything in mode-0.
  //   The tma_partition reorders and offsets mode-0 according to the tma_x atom and the multicast info.
  //

  auto [tAgA, tAsA] = tma_partition(tma_a, Int<0>{}, Layout<_1>{},
                                    group_modes<0,2>(sA), group_modes<0,2>(gA));  // (TMA,k) and (TMA,PIPE)

  auto [tBgB, tBsB] = tma_partition(tma_b, Int<0>{}, Layout<_1>{},
                                    group_modes<0,2>(sB), group_modes<0,2>(gB));  // (TMA,k) and (TMA,PIPE)

  // The TMA is responsible for copying everything in mode-0 of tAsA and tBsB
  constexpr int kTmaTransactionBytes = CUTE_STATIC_V(size<0>(tAsA)) * sizeof(TA) +
                                       CUTE_STATIC_V(size<0>(tBsB)) * sizeof(TB);

  //
  // PREFETCH
  //

  auto K_PIPE_MAX = size<1>(tAsA);

  // Total count of tiles
  int k_tile_count = size<1>(tAgA);
  // Current tile index in gmem to read from
  int k_tile = 0;

  // Initialize Barriers
  int warp_idx = cutlass::canonical_warp_idx_sync();
  int lane_predicate = cute::elect_one_sync();
  uint64_t* producer_mbar = smem.tma_barrier;
  uint64_t* consumer_mbar = smem.mma_barrier;

  using ProducerBarType = cutlass::arch::ClusterTransactionBarrier;  // TMA
  using ConsumerBarType = cutlass::arch::ClusterBarrier;             // MMA
  CUTE_UNROLL
  for (int pipe = 0; pipe < K_PIPE_MAX; ++pipe) {
    if ((warp_idx == 0) && lane_predicate) {
      ProducerBarType::init(&producer_mbar[pipe],   1);
      ConsumerBarType::init(&consumer_mbar[pipe], size(mma));
    }
  }
  // Ensure barrier init is complete on all CTAs
  cluster_sync();

  // Start async loads for all pipes
  CUTE_UNROLL
  for (int pipe = 0; pipe < K_PIPE_MAX; ++pipe)
  {
    if ((warp_idx == 0) && lane_predicate)
    {
      // Set expected Tx Bytes after each reset / init
      ProducerBarType::arrive_and_expect_tx(&producer_mbar[pipe], kTmaTransactionBytes);
      copy(tma_a.with(producer_mbar[pipe]), tAgA(_,k_tile), tAsA(_,pipe));
      copy(tma_b.with(producer_mbar[pipe]), tBgB(_,k_tile), tBsB(_,pipe));
    }
    --k_tile_count;
    ++k_tile;
  }

  //
  // Define A/B partitioning and C accumulators
  //
  // TUTORIAL:
  //   The tCrA and tCrB are actually Tensors of MMA Descriptors constructed as views of SMEM.
  //   The MMA Descriptor generation is automatic via inspection and validation of the SMEM Layouts.
  //   Because the MMA reads directly from SMEM and the fragments are descriptors rather than registers,
  //     there is no need for copy(tCsA, tCrA) in the mainloop.
  //

  ThrMMA thr_mma = mma.get_thread_slice(threadIdx.x);
  Tensor tCsA = thr_mma.partition_A(sA);                               // (MMA,MMA_M,MMA_K,PIPE)
  Tensor tCsB = thr_mma.partition_B(sB);                               // (MMA,MMA_N,MMA_K,PIPE)
//   Tensor tCgC = thr_mma.partition_C(gC);                               // (MMA,MMA_M,MMA_N)

  // Allocate accumulators and clear them
//   Tensor tCrC = thr_mma.make_fragment_C(tCgC);                         // (MMA,MMA_M,MMA_N)
  Tensor tCrC = partition_fragment_C(mma, select<0, 1>(cta_tiler));
  clear(tCrC);

  // Allocate descriptor iterators
  Tensor tCrA = thr_mma.make_fragment_A(tCsA);                         // (MMA,MMA_M,MMA_K,PIPE)
  Tensor tCrB = thr_mma.make_fragment_B(tCsB);                         // (MMA,MMA_N,MMA_K,PIPE)

  //
  // PIPELINED MAIN LOOP
  //
  // TUTORIAL:
  //   Rather than interleaving the stages and instructions like in SM70 and SM80,
  //     the SM90 mainloops rely on explicit producer-consumer synchronization
  //     on the purely async instructions TMA and MMA.
  //   More advanced pipeline and warp-specialization strategies are available in CUTLASS mainloops.
  //

  // A PipelineState is a circular pipe index [.index()] and a pipe phase [.phase()]
  //   that flips each cycle through K_PIPE_MAX.
  auto write_state = cutlass::PipelineState<K_PIPE_MAX>();             // TMA writes
  auto read_state  = cutlass::PipelineState<K_PIPE_MAX>();             // MMA  reads

  CUTE_NO_UNROLL
  while (k_tile_count > -K_PIPE_MAX)
  {
    // Wait for Producer to complete
    int read_pipe = read_state.index();
    ProducerBarType::wait(&producer_mbar[read_pipe], read_state.phase());

    // MMAs to cover 1 K_TILE
    warpgroup_arrive(); 
    gemm(mma, tCrA(_,_,_,read_pipe), tCrB(_,_,_,read_pipe), tCrC);     // (V,M) x (V,N) => (V,M,N)
    warpgroup_commit_batch();

    // Wait for all MMAs in a K_TILE to complete
    warpgroup_wait<0>();

    // Notify that consumption is done
    ConsumerBarType::arrive(&consumer_mbar[read_pipe]);
    ++read_state;

    if ((warp_idx == 0) && lane_predicate)
    {
      int pipe = write_state.index();
      // Wait for Consumer to complete consumption
      ConsumerBarType::wait(&consumer_mbar[pipe], write_state.phase());
      // Set expected Tx Bytes after each reset / init
      ProducerBarType::arrive_and_expect_tx(&producer_mbar[pipe], kTmaTransactionBytes);
      copy(tma_a.with(producer_mbar[pipe]), tAgA(_,k_tile), tAsA(_,pipe));
      copy(tma_b.with(producer_mbar[pipe]), tBgB(_,k_tile), tBsB(_,pipe));
      ++write_state;
    }
    --k_tile_count;
    ++k_tile;
  }

  //
  // Epilogue
  //

#if 1

    // Make sure all warpgroups have finished mma
    cutlass::arch::NamedBarrier::sync(size(mma), 0);

    using SmemCopyAtomC = Copy_Atom<cute::SM90_U32x4_STSM_N, TC>;
    
    Tensor sC = make_tensor(make_smem_ptr(smem.smem_C.data()), SmemLayoutC{}); // (BLK_M,BLK_N)

    auto smem_tiled_copy_C = make_tiled_copy_C(SmemCopyAtomC{}, mma);
    auto smem_thr_copy_C = smem_tiled_copy_C.get_thread_slice(threadIdx.x);
    // include this convert utility
    Tensor tCrC_out = convert_type<TC>(tCrC);
    Tensor taccCrC = smem_thr_copy_C.retile_S(tCrC_out);        // ((Atom,AtomNum), MMA_M, MMA_N)
    Tensor taccCsC = smem_thr_copy_C.partition_D(sC);     // ((Atom,AtomNum),PIPE_M,PIPE_N)
    cute::copy(smem_tiled_copy_C, taccCrC, taccCsC);
    cutlass::arch::fence_view_async_shared(); // ensure smem writes are visible to TMA

    // NumThreads == size(mma)
    cutlass::arch::NamedBarrier::arrive(size(mma) + cutlass::NumThreadsPerWarp,
        cutlass::arch::ReservedNamedBarriers::EpilogueBarrier);

    // Prepare TMA store

    // auto [M, N, K] = shape_MNK;
    Tensor mC = tma_store.get_tma_tensor(make_shape(M,N));
    Tensor gC = local_tile(mC, select<0,1>(cta_tiler), make_coord(blockIdx.x, blockIdx.y));
    auto block_tma_store = tma_store.get_slice(_0{}); // CTA slice
    Tensor tCgC = block_tma_store.partition_D(gC);  // (TMA, TMA_M, TMA_K)
    Tensor tCsC = block_tma_store.partition_S(sC);  // (TMA, TMA_M, TMA_K)

    // TMA STORE: SMEM -> GMEM
    // int const warp_idx = cutlass::canonical_warp_idx_sync();
    // int const lane_predicate = cute::elect_one_sync();
    if (warp_idx == 0) {
        // Ensure RMEM -> SMEM copy completes before issuing TMA store
        cutlass::arch::NamedBarrier::sync(
            size(mma) + cutlass::NumThreadsPerWarp, 
            cutlass::arch::ReservedNamedBarriers::EpilogueBarrier
        );
    }

    if (warp_idx == 0 && lane_predicate) {
        cute::copy(tma_store, tCsC, tCgC);
        // tma_store_arrive();
    }

    // tma_store_wait<0>();

#endif

}

// Setup params for a TN GEMM
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_tn(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
//   printf("gemm tn.\n");
  
  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define TN strides (mixed)
  auto dA = make_stride(ldA, Int<1>{});                      // (dM, dK)
  auto dB = make_stride(ldB, Int<1>{});                      // (dN, dK)
//   auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)
  auto dC = make_stride(ldC, Int<1>{});                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<256>{};
  auto bN = Int<192>{};
  auto bK = Int<128>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<2>{};  // Pipeline
  using AtomLayoutMNK = Layout<Shape<_2, _1, _1>>;

  // Define the smem layouts (static)
  auto sA = tile_to_shape(GMMA::Layout_K_SW128_Atom<TA>{}, make_shape(bM,bK,bP));
  auto sB = tile_to_shape(GMMA::Layout_K_SW128_Atom<TB>{}, make_shape(bN,bK,bP));

  // Define the Tiled MMA
  TiledMMA tiled_mma = make_tiled_mma(SM90_64x192x16_F32F16F16_SS<GMMA::Major::K,GMMA::Major::K>{},
    AtomLayoutMNK{});

  // For FP16 ACCUM example, enable these tile sizes and variable definitions.
  // auto bM = Int<256>{};
  // auto bN = Int<256>{};
  // auto bK = Int<96>{};
  // auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  // auto bP = Int<2>{};  // Pipeline
  // using AtomLayoutMNK = Layout<Shape<_4, _1, _1>>;

  // auto sA = tile_to_shape(GMMA::Layout_K_SW64_Atom<TA>{}, make_shape(bM,bK,bP));
  // auto sB = tile_to_shape(GMMA::Layout_K_SW64_Atom<TB>{}, make_shape(bN,bK,bP));

  // TiledMMA tiled_mma = make_tiled_mma(SM90_64x256x16_F16F16F16_SS<GMMA::Major::K,GMMA::Major::K>{},
  //   AtomLayoutMNK{});

  //
  // Back to common setting
  //
  
  // Define the smem layout for the output (static)
  auto sC = tile_to_shape(GMMA::Layout_K_SW128_Atom<TC>{}, make_shape(bM,bN));

  // Define the TMAs
  // Create Global memory tensors for TMA inspection
  Tensor mA = make_tensor(A, make_shape(M,K), dA);
  Tensor mB = make_tensor(B, make_shape(N,K), dB);
  Tensor mC = make_tensor(make_gmem_ptr(C), make_shape(M, N), dC);

  // Create TMA Atoms with the desired copy operation on the source and destination
  Copy_Atom tmaA = make_tma_atom(SM90_TMA_LOAD{}, mA, sA(_,_,0), make_shape(bM,bK));
  Copy_Atom tmaB = make_tma_atom(SM90_TMA_LOAD{}, mB, sB(_,_,0), make_shape(bN,bK));

  auto tma_store = make_tma_copy(SM90_TMA_STORE{}, mC, sC, make_shape(bM, bN), _1{});

  //
  // Setup and Launch
  //

  // Launch parameter setup
  int smem_size = int(sizeof(SharedStorage<TA, TB, TC, decltype(sA), decltype(sB), decltype(sC)>));

//   std::cout << "smem size is " << smem_size << std::endl;



  dim3 dimBlock(size(tiled_mma));
  // dim3 dimCluster(1, 1, 1);
  dim3 dimCluster(1, 2, 1);
  dim3 dimGrid(round_up(size(ceil_div(m, bM)), dimCluster.x),
               round_up(size(ceil_div(n, bN)), dimCluster.y));
  cutlass::ClusterLaunchParams params = {dimGrid, dimBlock, dimCluster, smem_size};

  void const* kernel_ptr = reinterpret_cast<void const*>(
                              &gemm_device<decltype(prob_shape), decltype(cta_tiler),
                                           TA, decltype(sA), decltype(tmaA),
                                           TB, decltype(sB), decltype(tmaB),
                                           TC, decltype(sC), decltype(tma_store),
                                           decltype(tiled_mma)>);

  CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
    kernel_ptr),
    hipFuncAttributeMaxDynamicSharedMemorySize,
    smem_size));

#if 1
  // Kernel Launch
  cutlass::Status status = cutlass::launch_kernel_on_cluster(params, kernel_ptr,
                                                             prob_shape, cta_tiler,
                                                             A, tmaA,
                                                             B, tmaB,
                                                             C, tma_store,
                                                             tiled_mma);
  CUTE_CHECK_LAST();


  if (status != cutlass::Status::kSuccess) {
    std::cerr << "Error: Failed at kernel Launch" << std::endl;
  }

#endif
}

template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm(char transA, char transB, int m, int n, int k,
     Alpha alpha,
     TA const* A, int ldA,
     TB const* B, int ldB,
     Beta beta,
     TC      * C, int ldC,
     hipStream_t stream = 0)
{
  if (transA == 'N' && transB == 'T') {
    // This example doesn't implement NT gemm for simplicity.
    // return gemm_nt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  } else
  if (transA == 'T' && transB == 'N') {
    return gemm_tn(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  }
  assert(false && "Not implemented");
}

int main(int argc, char** argv)
{

  hipDeviceProp_t props;
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (props.major != 9) {
    std::cout << "This example requires NVIDIA's Hopper Architecture GPU with compute capability 90a\n" << std::endl;
    return 0;
  }

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  int m = 8192;
  if (argc >= 2)
    sscanf(argv[1], "%d", &m);

  int n = 8192;
  if (argc >= 3)
    sscanf(argv[2], "%d", &n);

  int k = 8192;
  if (argc >= 4)
    sscanf(argv[3], "%d", &k);

  char transA = 'T';
//   if (argc >= 5)
//     sscanf(argv[4], "%c", &transA);

  char transB = 'N';
//   if (argc >= 6)
//     sscanf(argv[5], "%c", &transB);

  print("M N K = [%d %d %d].\n", m, n, k);

  using TA = cute::half_t;
  using TB = cute::half_t;
  using TC = cute::half_t;
  using TI = cute::half_t;

  TI alpha = TI(1.0f);
  TI beta  = TI(0.0f);

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  // Initialize the tensors
  // for (int j = 0; j < m*k; ++j) h_A[j] = TA(int((rand() % 2) ? 1 : -1));
  // for (int j = 0; j < n*k; ++j) h_B[j] = TB(int((rand() % 2) ? 1 : -1));
  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TB>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = TC(0);

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  double gflops = (2.0*m*n*k) * 1e-9;

  const int timing_iterations = 10;
  GPU_Clock timer;

  int ldA = 0, ldB = 0, ldC = n;

  if (transA == 'N') {
    ldA = m;
  } else if (transA == 'T') {
    ldA = k;
  } else {
    assert(false);
  }

  if (transB == 'N') {
    ldB = k;
  } else if (transB == 'T') {
    ldB = n;
  } else {
    assert(false);
  }

  // Run once
  d_C = h_C;
  gemm(transA, transB, m, n, k,
       alpha,
       d_A.data().get(), ldA,
       d_B.data().get(), ldB,
       beta,
       d_C.data().get(), ldC);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    gemm(transA, transB, m, n, k,
         alpha,
         d_A.data().get(), ldA,
         d_B.data().get(), ldB,
         beta,
         d_C.data().get(), ldC);
  }
  double cute_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUTE_GEMM:     [%6.1f]GFlop/s  (%6.4f)ms\n", gflops / cute_time, cute_time*1000);

#else

  std::cout << "CUTLASS_ARCH_MMA_SM90_SUPPORTED must be enabled, but it is not. Test is waived \n" << std::endl;
#endif

 return 0;

}
